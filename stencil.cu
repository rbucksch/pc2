#include "hip/hip_runtime.h"
/**
 * Projekt Parallel Computing 2
 * Waermeleitung auf NxN-Gitter unter Verwendung von Cuda
 *
 * Autor: Robert Bucksch
 * Datum: 18.06.19
 */
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// TDIM x TDIM Threads pro Block
#define TDIM 16 

// vertauscht zwei Pointer
__host__ __device__ void swap(double *&a, double *&b) {
  double* tmp = a;
  a = b;
  b = tmp;
}

// speichert Grenzen des Shared-Memory-Bereichs, verschiebt diese
// nach innen, falls Block einen Bereich am Rand des Gitters bearbeitet
struct bounds {
  __device__ bounds(int smdim, int gc) {
    top    = (blockIdx.y == 0 ? gc : 0);
    left   = (blockIdx.x == 0 ? gc : 0);
    right  = smdim - (blockIdx.x == gridDim.x - 1 ? gc : 0);
    bottom = smdim - (blockIdx.y == gridDim.y - 1 ? gc : 0);
  }
  int top, left, right, bottom;
};

// befuellt die beiden Zonen sm und tmp des Shared Memory mit Daten aus globalem Array t1
__device__ void fillSM(double *sm, double *gm, int smdim, int gmdim, int gc,
                       bounds b) {
  // Offset von Shared Memory bzgl globalem Array gm berechnen
  int gOS = blockDim.x * (blockDim.y * blockIdx.y * gridDim.x+ blockIdx.x)
            - gc * gmdim - gc;

  for (int i = threadIdx.y + b.top; i < b.bottom; i += blockDim.y) {
    for (int j = threadIdx.x + b.left; j < b.right; j += blockDim.x) {
      double s = gm[gOS + i * gmdim + j];
      // schreibe in sm
      sm[i * smdim + j] = s;
      // schreibe in tmp
      sm[i * smdim + j + smdim * smdim] = s;
    }
  }
}

// schreibt die finalen Daten im Shared Memory an die entsprechenden Stellen in t2
__device__ void write_back(double *sm, double *gm, int smdim, int gmdim, int gc) {
  // Offset in global memory berechnen
  int gOS = blockDim.x * (blockDim.y * blockIdx.y * gridDim.x+ blockIdx.x);
  // Offset in shared memory berechnen
  int sOS = gc * smdim + gc;

  gm[gOS + threadIdx.y * gmdim + threadIdx.x] = sm[sOS + threadIdx.y * smdim + threadIdx.x];
}

__device__ void update_sm(double *sm, double *tmp, int smdim, bounds b) {
  for (int i = threadIdx.y + b.top + 1; i < b.bottom - 1; i += blockDim.y) {
    for (int j = threadIdx.x + b.left + 1; j < b.right - 1; j += blockDim.x) {
      tmp[i * smdim + j] = 0.2 * (  sm[(i - 1) * smdim + j]
                                  + sm[i * smdim + (j - 1)]
                                  + sm[i * smdim + j]
                                  + sm[i * smdim + (j + 1)]
                                  + sm[(i + 1) * smdim + j]);
    }
  }
}

__global__ void update(double *t1, double *t2, int size, int gc, int smdim) {
  extern __shared__ double shared[];

  // teile Shared Memory in zwei Bereiche auf
  double* sm = &shared[0];
  double* tmp = &shared[smdim*smdim];

  bounds b(smdim, gc);

  // shared memory befuellen
  fillSM(shared, t1, smdim, size, gc, b);
  __syncthreads();

  // sm so oft updaten, wie es die Breite der Geisterzone erlaubt
  for (int k = 0; k < gc; ++k) {
    update_sm(sm, tmp, smdim, b);
    __syncthreads();
    swap(sm, tmp);
  }

  // Daten zurueckschreiben
  write_back(sm, t1, smdim, size, gc);
}

// Initialisiere Wärmefeld mit Startwerten:
// innen: 0.0
// Rand:
// links/oben warm=25.0
// rechts/unten kalt=-25.0
void init(double *t, int size) {
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      t[j + i * size] = 0.0;
      if (j == 0)
        t[i * size] = 25.0;
      if (j == size - 1)
        t[j + i * size] = -25.0;
      if (i == 0)
        t[j + i * size] = 25.0;
      if (i == size - 1)
        t[j + i * size] = -25.0;
    }
  }
}

// Ausgabe des Feldes t als PPM (Portable Pix Map) in filename
// mit schönen Farben
void printResult(double *t, int size, char *filename) {
  FILE *f = fopen(filename, "w");
  fprintf(f, "P3\n%i %i\n255\n", size, size);
  double tmax = 25.0;
  double tmin = -tmax;
  double r, g, b;
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      double val = t[j + i * size];
      r = 0;
      g = 0;
      b = 0;
      if (val <= tmin) {
        b = 1.0 * 255.0;
      } else if (val >= -25.0 && val < -5) {
        b = 255 * 1.0;
        g = 255 * ((val + 25) / 20);
      } else if (val >= -5 && val <= 0.0) {
        g = 255 * 1.0;
        b = 255 * (1.0 - (val + 5) / 5);
      } else if (val > 0.0 && val <= 5) {
        g = 255 * 1.0;
        r = 255 * ((val) / 5);
      } else if (val > 5 && val < 25.0) {
        r = 255 * 1.0;
        g = 255 * ((25 - val) / 20);
      } else {
        r = 255 * 1.0;
      }
      fprintf(f, "%i\n%i\n%i\n", (int)r, (int)g, (int)b);
    }
    //      fprintf(f,"\n");
  }
  fclose(f);
}

int main(int argc, char **argv) {
  // Größe des Feldes
  int size = 128;
  // Breite der Geisterzone
  int ghostcells = 2;
  // Anzahl Iterationen
  int iter = 100;
  // Ausgabedatei
  char *filename = (char*)"out.ppm";

  // Übergabeparameter für [size ghostcells iter filename] einlesen
  if (argc > 1) size = atoi(argv[1]);
  if (argc > 2) ghostcells = atoi(argv[2]);
  if (argc > 3) iter = atoi(argv[3]);
  if (argc > 4) filename = argv[4];

  if (ghostcells < 1) {
    printf("Error: minimal width of ghostzone is 1\n");
    printf("Usage: %s [size] [ghostcells] [iter] [filename]\n", argv[0]);
    return -1;
  } else if (size % TDIM != 0) {
    printf("Error: size must be a multiple of %d\n", TDIM);
    printf("Usage: %s [size] [ghostcells] [iter] [filename]\n", argv[0]);
    return -1;
  }

  // Cache Config
  hipDeviceSetCacheConfig(hipFuncCachePreferNone);

  // 2 Speicherbereiche für das Wärmefeld auf Host und Device
  double *t1_host, *t2_host;
  double *t1_dev, *t2_dev;

  // Größe des Speicherbereiches
  int mem = size * size * sizeof(double);
  // Allokiere Speicher auf Host
  t1_host = (double *)malloc(mem);
  t2_host = (double *)malloc(mem);
  // Initialisiere Speicher
  init(t1_host, size);

  // CUDA Speicher anlegen
  hipMalloc((void **)&t1_dev, mem);
  hipMalloc((void **)&t2_dev, mem);

  // Host->Device Memcpy von t1
  hipMemcpy(t1_dev, t1_host, mem, hipMemcpyHostToDevice);

  // 2D Threads/Grid anlegen
  int tdim = min(TDIM, size);
  dim3 threads(tdim, tdim);
  dim3 grid(size / threads.x, size / threads.y);

  // Wiederhole update Kernel für iter Iterationen
  for (int iters_left = iter; iters_left > 0; iters_left -= ghostcells) {
    int gc = min(iters_left, ghostcells);
    int smdim = tdim + 2 * gc;
    int smem = smdim * smdim * sizeof(double);
    update<<<grid, threads, 2*smem>>>(t1_dev, t2_dev, size, gc, smdim);
  }

  // Kopiere Endzustand zurück
  hipMemcpy(t2_host, t1_dev, mem, hipMemcpyDeviceToHost);

  // Ausgabe des Endzustandes mit printResult
  printResult(t2_host, size, filename);

  // Speicher Freigeben (Device+Host)
  free(t1_host);
  free(t2_host);
  hipFree(t1_dev);
  hipFree(t2_dev);

  return 0;
}
